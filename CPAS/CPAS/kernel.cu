#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_complex.h"
#include "math_functions.h"
#include "hip/hip_math_constants.h":
#include <hipfft/hipfft.h>

#include <stdio.h>

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}

__device__ __forceinline__ hipComplex expf(hipComplex z)
{

	hipComplex res;
	float t = expf(z.x);
	sincosf(z.y, &res.y, &res.x);
	res.x *= t;
	res.y *= t;

	return res;

}

__global__ void calculate(hipComplex *fths, int *xo, int *yo, int *uo, float *zo2, float dfxs,  float lambda, float k0, int Ts, float *fxs, float * y0seg, float* x0seg, int S_Bx, int S_By, int N_Bx, int N_By)
{
	
//	int	index = blockIdx.y*(blockDim.x*gridDim.x) + blockIdx.x*blockDim.x + threadIdx.x;
	//	int index = threadIdx.x*blockIdx.z + blockIdx.y*blockDim.z + blockIdx.x*blockDim.x;


	//	int blockId = blockIdx.x + blockIdx.y * gridDim.x  + gridDim.x * gridDim.y * blockIdx.z;
	//	int threadId = blockId * blockDim.x + threadIdx.x;

	//	int pt_indx = threadIdx.x*blockIdx.z;


	float yp = yo[threadIdx.x] - y0seg[blockIdx.y];
	
	float xp = xo[threadIdx.x] - x0seg[blockIdx.x];

	float rp = sqrt(zo2[threadIdx.x ] + xp*xp + yp*yp);
	

	float inv_rp = 1 / rp;

	float fxp = xp*inv_rp / lambda;
	float fyp = yp*inv_rp / lambda;



	int iifx = round(fxp / dfxs) + S_Bx / 2 + 1;
	int iify = round(fyp / dfxs) + S_By / 2 + 1;

	if (iifx <= 0 || iifx > S_Bx || iify <= 0 || iify > S_Bx){
		iifx = S_Bx / 2 + 1;
		iify = S_Bx / 2 + 1;
	}
	


	hipComplex c0;
	hipComplex arg;
	arg.x = (k0*rp - 2 * HIP_PI_F*(fxs[iifx] + fxs[iify])*(Ts / 2)*inv_rp);
	
	c0 = expf(arg);
//	c0.x = uo[blockDim.x] * c0.x;
//	c0.y = uo[blockDim.x] * c0.y;

	//fths[threadId] = c0;
//	Nep[threadId] = iifx;
//	Nip[threadId] = iify;

	fths[iifx + blockIdx.x*S_Bx + iify*S_Bx*N_Bx + blockIdx.x* S_Bx*N_Bx*S_By].x += c0.x;
	fths[iifx + blockIdx.x*S_Bx + iify*S_Bx*N_Bx + blockIdx.x* S_Bx*N_Bx*S_By].y += c0.y;
}

hipfftResult preparePlan2D(hipfftHandle* plan, int nRows, int nCols, int batch){

	int n[2] = { nRows, nCols };

	hipfftResult result = hipfftPlanMany(plan,
		2, //rank
		n, //dimensions = {nRows, nCols}
		0, //inembed
		batch, //istride
		1, //idist
		0, //onembed
		batch, //ostride
		1, //odist
		HIPFFT_C2C, //hipfftType
		batch /*batch*/);

	if (result != 0){

//		std::cout << "preparePlan2D error, result: " << result << "/n";
		return result;
	}
	return result;
}

hipfftResult execute2D(hipfftHandle* plan, hipfftComplex* idata, hipfftComplex* odata, int direction){

	hipfftResult result = hipfftExecC2C(*plan, idata, odata, direction);

	if (result != 0){

//		cout << "execute2D error, result: " << result << "/n";
		return result;
	}
	return result;
}


__global__ void copy2bitmap(hipComplex *H, int *bitmap_H)
{

}


__global__ void asemble(hipComplex *fths, int *xo, int *yo, int *uo, float *zo2, float *dfxs, int *Nxs, float *lambda, int *Ts, float * fxs, float* y0seg, float* x0seg, int* Nep, int* Nip)
{
	//int	index = blockIdx.y*(blockDim.x*gridDim.x) + blockIdx.x*blockDim.x + threadIdx.x;
	//	int index = threadIdx.x*blockIdx.z + blockIdx.y*blockDim.z + blockIdx.x*blockDim.x;


	int blockId = blockIdx.x + blockIdx.y * gridDim.x + gridDim.x * gridDim.y * blockIdx.z;
	int threadId = blockId * blockDim.x + threadIdx.x;

	int pt_indx = threadIdx.x*blockIdx.z;

//	fths2[] = fths2[] + fths[threadId]

//		fths[threadId] = c0;
//	Nep[threadId] = iifx;
//	Nip[threadId] = iify;


}


void CPAS_CGH_3DPS_2d(int Np, int* xo, int* yo, int* zo, int* uo, int Nx, int Ny, int dx, float lambda, int S_Bx, int S_By, hipComplex* fths_p)
{
	hipEvent_t start, stop;
	float time;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	double k0 = 2 * HIP_PI_F / lambda;

	int x_size = (Nx / 2) + ((Nx / 2) - 1) + 1;
	int y_size = (Ny / 2) + ((Ny / 2) - 1) + 1;

	float *x = (float*) malloc(x_size * sizeof(float));
	float *y = (float*) malloc(y_size * sizeof(float));

	for (int t = 0; t < x_size; t++){
		x[t] = (-Nx / 2 + t)*dx;
	}

	for (int t = 0; t < y_size; t++){
		y[t] = (-Ny / 2 + t)*dx;
	}

	int N_Bx = Nx / S_Bx; // doda� obs�ug� nie ca�kowitych dziele�
	int N_By = Ny / S_By;

	int Ts = S_Bx*dx;
	
	float dfxs = 1 / (float)Ts;

	int fxs_size = (S_Bx / 2) + ((S_Bx / 2) - 1) + 1;
	float *fxs = (float*)malloc(fxs_size * sizeof(float));

	for (int t = 0; t < fxs_size; t++){
		fxs[t] = (float)(-S_Bx / 2 + t)*dfxs;
	}

	float * x0seg = (float*)malloc((N_Bx)* sizeof(float));

	for (int t = 0; t < N_By; t++){
		x0seg[t] = (x[0] + (t*Ts) + Ts / 2);
	}

	float * y0seg = (float*)malloc((N_By)* sizeof(float));

	for (int t = 0; t < N_By; t++){
		y0seg[t] = (y[0] + (t*Ts) + Ts / 2);
	}
/*	float * nseg_bx = (float*)malloc((Nosx)* sizeof(float));

	for (int t = 0; t < Nosx; t++){
		nseg_bx[t] = (1 + (t*Nxs));

	}

	float * nseg_by = (float*)malloc((Nosy)* sizeof(float));

	for (int t = 0; t < Nosy; t++){
		nseg_by[t] = (1 + (t*Nxs));
	}

	float *h = (float*)calloc(Nx, sizeof(float));
*/

	float *z02;
	z02 = (float*)malloc((Np)* sizeof(float));

	for (int t = 0; t < Np; t++)
		z02[t] = zo[t] * zo[t];



//	hipMalloc(&fths_p, sizeof(hipComplex)*N_Bx*N_By*S_Bx*S_By);
//	hipMemset(fths_p, 0, sizeof(hipComplex)*N_Bx*N_By*S_Bx*S_By);


	int *d_xo;
	int *d_yo;
	float *d_z02;
	
	hipMalloc((void**)&d_xo, sizeof(int)*Np);
	hipMalloc((void**)&d_yo, sizeof(int)*Np);
	hipMalloc((void**)&d_z02, sizeof(float)*Np);

	hipMemcpy(d_xo, xo, Np*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_yo, yo, Np*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_z02, z02, Np*sizeof(float), hipMemcpyHostToDevice);

	float *d_fxs;
	float *d_y0seg;
	float *d_x0seg;

	hipMalloc((void**)&d_x0seg, sizeof(float)*N_Bx);
	hipMalloc((void**)&d_y0seg, sizeof(float)*N_By);
	hipMalloc((void**)&d_fxs, sizeof(float)*fxs_size);

	hipMemcpy(d_fxs, fxs, fxs_size*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_x0seg, x0seg, N_Bx*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_y0seg, y0seg, N_By*sizeof(float), hipMemcpyHostToDevice);

	dim3 grid;
	grid.x = N_Bx;//y
	grid.y = N_By;//x

	dim3 block;
	block.x = Np; //z
	block.y = 1;
	
	hipEventRecord(start, 0);
	calculate << < grid, block >> >(fths_p, d_xo, d_yo, uo, d_z02, dfxs, lambda, k0, Ts, d_fxs, d_y0seg, d_x0seg, S_Bx, S_Bx, N_Bx, N_By);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	printf("Time for the kernel: %f ms\n", time);

/*	hipComplex *host;
	host = (hipComplex*)malloc(sizeof(hipComplex)*Nosx*Nosy*Np);
	hipMemcpy(host, fths, sizeof(hipComplex)*Nosx*Nosy*Np, hipMemcpyDeviceToHost);
*/
}

int main()
{




	hipEvent_t start, stop;
	float time;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	/*START CUDA CALC PART - DEKLARACJE*/
	int Nx = 1024;
	int Ny = 1024;
	int dx = 8;
	float lambda = 0.5;
	/*START CUDA FFT 2D PART - DEKLARACJE*/
	int S_Bx = 8; 
	int S_By = 8;

	hipfftComplex* h_out; //dane wynikowe CPU
	hipfftComplex* holo; //dane wyj�ciowe GPU

	int batch = Nx/S_Bx * Ny/S_By;  //N_Bx*N_By
	hipfftHandle forwardPlan;

	preparePlan2D(&forwardPlan, S_Bx, S_By, batch);
	


	h_out = (hipfftComplex*)malloc(sizeof(hipfftComplex)*S_Bx*S_By*batch); //allokacja pami�ci na wynik (CPU)


	
	hipMalloc(&holo, sizeof(hipfftComplex) *S_Bx*S_By*batch); //allokacja pami�ci na dane wyj�ciowe (GPU)
	hipMemset(holo, 0, sizeof(hipfftComplex)*S_Bx*S_By*batch); //Wype�nianie zaalokowanej pami�ci zerami (GPU)

	/*END CUDA FFT 2D PART - DEKLARACJE*/

	/*Kod kernela*/
	int Np = 1000;

	int *xo;
	int *yo;
	int *zo;
	int *uo;
	

	xo = (int*)malloc((Np)* sizeof(int));
	yo = (int*)malloc((Np)* sizeof(int));
	zo = (int*)malloc((Np)* sizeof(int));
	uo = (int*)malloc((Np)* sizeof(int));

	for (int tt = 0; tt < Np; tt++)
	{
		xo[tt] = tt;
		yo[tt] = Np - tt;
		zo[tt] = yo[tt] * xo[tt];
	}


	
	hipComplex *fths_p;

	hipfftComplex* fhs; 

//	hipMalloc(&fhs, sizeof(hipfftComplex)*S_Bx*S_By*batch); //allokacja pami�ci na dane wej�ciowe (GPU)
	hipMalloc(&fths_p, sizeof(hipComplex)*Nx*Ny);
	hipMemset(fths_p, 0, sizeof(hipComplex)*Nx*Ny);

	hipEventRecord(start, 0);
	/*START CUDA CALC PART */
	CPAS_CGH_3DPS_2d(Np, xo, yo, zo, uo, Nx, Ny, dx, lambda, S_Bx, S_By, fths_p);

	/*START CUDA CALC PART */

	
	


	/*START CUDA FFT PART */
	execute2D(&forwardPlan, fths_p, holo, HIPFFT_FORWARD); 

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	/*Wyswietlanie modulu/fazy*/
	hipMemcpy(h_out, holo, sizeof(hipfftComplex)*S_Bx*S_By*batch, hipMemcpyDeviceToHost); 

	
	/*END CUDA FFT PART */





	// Retrieve result from device and store it in host array
	hipEventElapsedTime(&time, start, stop);
	printf("Time for the kernel: %f ms\n", time);
	printf("Time for the kernel: %f ms\n", time);

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
/*	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}
	*/
	return 0;
}

