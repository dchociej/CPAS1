#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_complex.h"
#include "math_functions.h"
#include "hip/hip_math_constants.h":
#include <hipfft/hipfft.h>

#include <stdio.h>

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}

__device__ __forceinline__ hipComplex expf(hipComplex z)
{

	hipComplex res;
	float t = expf(z.x);
	sincosf(z.y, &res.y, &res.x);
	res.x *= t;
	res.y *= t;

	return res;

}

__global__ void calculate(hipComplex *fths, int *xo, int *yo, int *uo, float *zo2, float *dfxs, int *Nxs, float *lambda, int *Ts, float * fxs, float* y0seg, float* x0seg, int S_Bx, int S_By, int N_Bx, int N_By)
{
	int	index = blockIdx.y*(blockDim.x*gridDim.x) + blockIdx.x*blockDim.x + threadIdx.x;
	//	int index = threadIdx.x*blockIdx.z + blockIdx.y*blockDim.z + blockIdx.x*blockDim.x;


	//	int blockId = blockIdx.x + blockIdx.y * gridDim.x  + gridDim.x * gridDim.y * blockIdx.z;
	//	int threadId = blockId * blockDim.x + threadIdx.x;

	//	int pt_indx = threadIdx.x*blockIdx.z;

	float yp = yo[threadIdx.x] - y0seg[blockIdx.y];
	float xp = xo[threadIdx.x] - x0seg[blockIdx.x];

	float rp = sqrt(zo2[blockIdx.x] + xp*xp + yp*yp);


	float inv_rp = 1 / rp;

	float fxp = xp*inv_rp / *lambda;
	float fyp = yp*inv_rp / *lambda;

	float k0 = 2 * HIP_PI_F / *lambda;

	int iifx = round(fxp / *dfxs) + *Nxs / 2 + 1;
	int iify = round(fyp / *dfxs) + *Nxs / 2 + 1;

	if (iifx <= 0 || iifx > *Nxs || iify <= 0 || iify > *Nxs){
		iifx = *Nxs / 2 + 1;
		iify = *Nxs / 2 + 1;
	}

	hipComplex c0;
	hipComplex arg;
	arg.x = (k0*rp - 2 * HIP_PI_F*(fxs[iifx] + fxs[iify])*(*Ts / 2)*inv_rp);

	c0 = expf(arg);
	c0.x = uo[blockDim.x] * c0.x;
	c0.y = uo[blockDim.x] * c0.y;

	//fths[threadId] = c0;
//	Nep[threadId] = iifx;
//	Nip[threadId] = iify;

//	fths[iifx + blockIdx.x*S_Bx + iify*S_Bx*N_Bx + blockIdx.x* S_Bx*N_Bx*S_By].x += c0.x;

//	fths[iifx + blockIdx.x*S_Bx + iify*S_Bx*N_Bx + blockIdx.x* S_Bx*N_Bx*S_By].y += c0.y;
}

hipfftResult preparePlan2D(hipfftHandle* plan, int nRows, int nCols, int batch){

	int n[2] = { nRows, nCols };

	hipfftResult result = hipfftPlanMany(plan,
		2, //rank
		n, //dimensions = {nRows, nCols}
		0, //inembed
		batch, //istride
		1, //idist
		0, //onembed
		batch, //ostride
		1, //odist
		HIPFFT_C2C, //hipfftType
		batch /*batch*/);

	if (result != 0){

//		std::cout << "preparePlan2D error, result: " << result << "/n";
		return result;
	}
	return result;
}

hipfftResult execute2D(hipfftHandle* plan, hipfftComplex* idata, hipfftComplex* odata, int direction){

	hipfftResult result = hipfftExecC2C(*plan, idata, odata, direction);

	if (result != 0){

//		cout << "execute2D error, result: " << result << "/n";
		return result;
	}
	return result;
}


__global__ void copy2bitmap(hipComplex *H, int *bitmap_H)
{

}


__global__ void asemble(hipComplex *fths, int *xo, int *yo, int *uo, float *zo2, float *dfxs, int *Nxs, float *lambda, int *Ts, float * fxs, float* y0seg, float* x0seg, int* Nep, int* Nip)
{
	//int	index = blockIdx.y*(blockDim.x*gridDim.x) + blockIdx.x*blockDim.x + threadIdx.x;
	//	int index = threadIdx.x*blockIdx.z + blockIdx.y*blockDim.z + blockIdx.x*blockDim.x;


	int blockId = blockIdx.x + blockIdx.y * gridDim.x + gridDim.x * gridDim.y * blockIdx.z;
	int threadId = blockId * blockDim.x + threadIdx.x;

	int pt_indx = threadIdx.x*blockIdx.z;

//	fths2[] = fths2[] + fths[threadId]

//		fths[threadId] = c0;
//	Nep[threadId] = iifx;
//	Nip[threadId] = iify;


}


void CPAS_CGH_3DPS_2d(int* xo, int Np, int* yo, int yo_size, int* zo, int zo_size, int* uo, int Nx, int Ny, int dx, float lambda, int Nxs)
{



	int x_size = (Nx / 2) + ((Nx / 2) - 1) + 1;
	int y_size = (Ny / 2) + ((Ny / 2) - 1) + 1;

	float *x = (float*)malloc(x_size * sizeof(float));
	float *y = (float*)malloc(y_size * sizeof(float));

	for (int t = 0; t < x_size; t++){
		x[t] = (-Nx / 2 + t)*dx;
	}

	for (int t = 0; t < y_size; t++){
		y[t] = (-Ny / 2 + t)*dx;
	}

	int Nosx = Nx / Nxs;
	int Nosy = Ny / Nxs;

	int Ts = Nxs*dx;
	float dfxs = 1 / (float)Ts;

	int fxs_size = (Nxs / 2) + ((Nxs / 2) - 1) + 1;
	float *fxs = (float*)malloc(fxs_size * sizeof(float));

	for (int t = 0; t < fxs_size; t++){
		fxs[t] = (float)(-Nxs / 2 + t)*dfxs;
	}

	float * x0seg = (float*)malloc((Nosx)* sizeof(float));

	for (int t = 0; t < Nosx; t++){
		x0seg[t] = (x[0] + (t*Ts) + Ts / 2);
	}

	float * y0seg = (float*)malloc((Nosy)* sizeof(float));

	for (int t = 0; t < Nosy; t++){
		y0seg[t] = (y[0] + (t*Ts) + Ts / 2);
	}

	float * nseg_bx = (float*)malloc((Nosx)* sizeof(float));

	for (int t = 0; t < Nosx; t++){
		nseg_bx[t] = (1 + (t*Nxs));

	}

	float * nseg_by = (float*)malloc((Nosy)* sizeof(float));

	for (int t = 0; t < Nosy; t++){
		nseg_by[t] = (1 + (t*Nxs));
	}

	float *h = (float*)calloc(Nx, sizeof(float));
	float zo2 = zo[0] * zo[0];

	hipEvent_t start, stop;
	float time;
	hipEventCreate(&start);
	hipEventCreate(&stop);


	//fths[x + (y*Nxs) + (z*Nxs*Nosx)];

	hipComplex *fths;

	hipMalloc(&fths, sizeof(hipComplex)*Nosx*Nosy*Nxs*Nxs);
	hipMemset(fths, 0, sizeof(hipComplex)*Nxs*Nxs*Nosx*Nosy);

	dim3 grid;
	grid.x = Nosx;//y
	grid.y = Nosy;//x

	dim3 block;
	block.x = Np; //z
	block.y = 1;
	
	hipEventRecord(start, 0);
	calculate <<< grid, block >>>(fths, xo, yo, uo, &zo2, &dfxs, &Nxs, &lambda, &Ts, fxs, y0seg, x0seg, 16, 16 ,128 ,128);

	hipComplex *host;
	host = (hipComplex*)malloc(sizeof(hipComplex)*Nosx*Nosy*Np);
	hipMemcpy(host, fths, sizeof(hipComplex)*Nosx*Nosy*Np, hipMemcpyDeviceToHost);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	printf("Time for the kernel: %f ms\n", time);

}

int main()
{
	const int arraySize = 5;
	const int a[arraySize] = { 1, 2, 3, 4, 5 };
	const int b[arraySize] = { 10, 20, 30, 40, 50 };
	int c[arraySize] = { 0 };

	hipEvent_t start, stop;
	float time;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// Add vectors in parallel.
	hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addWithCuda failed!");
		return 1;
	}

	printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
		c[0], c[1], c[2], c[3], c[4]);

	int nRows = 16;
	int nCols = 16;

	hipfftComplex* h_in; //dane wej�ciowe CPU
	hipfftComplex* h_out; //dane wynikowe CPU
	hipfftComplex* d_in; //dane wej�ciowe GPU
	hipfftComplex* d_out; //dane wyj�ciowe GPU

	int batch = 128 * 128;
	hipfftHandle forwardPlan;


	hipEventRecord(start, 0);

	preparePlan2D(&forwardPlan, nRows, nCols, batch);

//	h_in = convertMatToCufftComplex(&image, nCols, nRows, false); //konwersja obrazu do typu hipfftComplex (CPU)
	h_out = (hipfftComplex*)malloc(sizeof(hipfftComplex)*nRows*nCols*batch); //allokacja pami�ci na wynik (CPU)
	h_in = (hipfftComplex*)malloc(sizeof(hipfftComplex)*nRows*nCols*batch); //allokacja pami�ci na wynik (CPU)

	hipMalloc(&d_in, sizeof(hipfftComplex)*nRows*nCols*batch); //allokacja pami�ci na dane wej�ciowe (GPU)
		
	hipEventRecord(start, 0);
		
		hipMemcpy(d_in, h_in, sizeof(hipfftComplex)*nRows*nCols*batch, hipMemcpyHostToDevice); //kopiowanie danych wej�ciowych na GPU

	hipMalloc(&d_out, sizeof(hipfftComplex)*nRows*nCols*batch); //allokacja pami�ci na dane wyj�ciowe (GPU)
	hipMemset(d_out, 0, sizeof(hipfftComplex)*nRows*nCols*batch); //Wype�nianie zaalokowanej pami�ci zerami (GPU)

	/*Kod kernela*/
	int xo_size = 10;
	int yo_size = 10;
	int zo_size = 10;
	int uo_size = 10;

	int *xo;
	int *yo;
	int *zo;
	int *uo;

	xo = (int*)malloc((xo_size)* sizeof(int));
	yo = (int*)malloc((yo_size)* sizeof(int));
	zo = (int*)malloc((zo_size)* sizeof(int));
	uo = (int*)malloc((uo_size)* sizeof(int));
	
	int Nx = 2048;
	int Ny = 2048;
	int dx = 8;
	float lambda = 0.5;
	float Nsx = 16;

	CPAS_CGH_3DPS_2d(xo, xo_size, yo, yo_size, zo, zo_size,uo, Nx, Ny, dx,lambda,Nsx);

	/*Koniec*/
hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	execute2D(&forwardPlan, d_in, d_out, HIPFFT_FORWARD); //Policzenie FFT
	
	
	
	hipMemcpy(h_out, d_out, sizeof(hipfftComplex)*nRows*nCols*batch, hipMemcpyDeviceToHost); //Kopiowanie wyniku do pami�ci CPU

	//h_out wynik zawieraj�cy cz�� rzeczywist� i urojon�






	// Retrieve result from device and store it in host array
	hipEventElapsedTime(&time, start, stop);
	printf("Time for the kernel: %f ms\n", time);

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
	int *dev_a = 0;
	int *dev_b = 0;
	int *dev_c = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Launch a kernel on the GPU with one thread for each element.
	addKernel << <1, size >> >(dev_c, dev_a, dev_b);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_c);
	hipFree(dev_a);
	hipFree(dev_b);

	return cudaStatus;
}
